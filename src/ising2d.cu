/*
Ising model: Halmitonian H = /sum_ij J(sigma_i)(sigma_j)
*/

/*
* TODO:
*   1. Calculate the energy in the program
*   2. Calculate the heat capacity in the program
*   3. Add more inputs to adjust the length of lattice
*   4. A matlab code to plot data.
*       data format example:
*                    position.x  position.y   spin(-1, 1)
*       Iteattion 1:    1           4               -1
*                       *           *                *
*                       *           *                *
*       Iteattion 2:    4           3                1
*                       *           *                *
*                       *           *                *
*       Iteattion N:    35          76               1
*                       *           *                *
*                       *           *                *
*   5. Compare the numerical value with the analytic value
*   6. Move to 3D
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>       /* time */
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

/*
* LATTICE_LENGTH is the length of the lattice
* LATTICE_LENGTH is the number of element is one lattice
* BOLTZMANN_CONST is bolzmann constant. It is set to 1.
*/

#define  LATTICE_LENGTH 256
#define  LATTICE_2 (LATTICE_LENGTH * LATTICE_LENGTH)
#define  BOLTZMANN_CONST 1
#define  N LATTICE_LENGTH
#define  TIME_LENGTH 1e6

__global__ void printstate(double *energy);
__device__ double local_energy(int up, int down, int left, int right, int center);
__global__ void updateEnergy(int* lattice, double* energy, int init);
__global__ void update_random(int* lattice, double* random, const unsigned int offset, double beta);
__global__ void update(int* lattice, const unsigned int offset, double beta, hiprandState* state);

__global__ void ini_rng(hiprandState *state, unsigned long seed);


__global__ void ini_rng(hiprandState *state, unsigned long seed){
    const unsigned int idx = blockIdx.x * blockDim.y + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    hiprand_init(seed, idx + idy * N, 0, &state[idx + idy * N]);
}


/*
*   update is the function to update a point
*   1. flip a point (1 -> -1 or -1 -> 1)
*   2. compare the energy before flip a point and after flip a point
*   3. if the energy with flipped point is small, accept
*   4. if the energy is larger, generate a random number pro_rand (0,1),
*      if pro_rand < e^(-beta * delatE), aceept. else reject.
*/

__global__ void update(int* lattice, const unsigned int offset, double beta, hiprandState* state){
    // Calculate the global index
    // Calculate the global index for the up, down, left, right index.
    const unsigned int idx = blockIdx.x * blockDim.y + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int idx_l = (idx - 1 + N) % N;
    const unsigned int idx_r = (idx + 1 + N) % N;
    const unsigned int idy_u = (idy - 1 + N) % N;
    const unsigned int idy_d = (idy + 1 + N) % N;
    int flip, up, down, left, right, center;
    double pro_rand;
    double deltaE;

    // To generate random number in cuda
    hiprandState local_state = state[idx + idy * N];
    pro_rand = hiprand_uniform(&local_state);
    state[idx + idy * N] = local_state;

    if (idx < N && idy < N && idx_l < N && idx_r < N && idy_u < N && idy_d < N){
        if( ((idx + idy) % 2 == 0 && offset == 0) || ((idx + idy) % 2 == 1 && offset == 1) ){

            up = lattice[idx + idy_u * N];
            down = lattice[idx + idy_d * N];
            left = lattice[idx_l + idy * N];
            right = lattice[idx_r + idy * N];
            center = lattice[idx + idy * N];

            // Flip the center element
            flip = -center;
            // Calculate the difference between these two state
            deltaE = local_energy(up, down, left, right, flip);
            deltaE -= local_energy(up, down, left, right, center);

            // If deltaE < 0 or pro_rand <= e^(-beta * deltaE), accept new value
            if (pro_rand <= exp(- beta * deltaE)){
                lattice[idx + idy * N ] = flip;
            }
        }
    }
}

/*
*   printstate is the function to print the whole matrix.
*   Since it prints in parallel, we also print the global
*   index of the matrx.
*   it prints (x, y, (1 or -1)).
*/
__global__ void printstate(double* energy) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < N && idy < N){
        printf("%d, %d, %f\n", idx, idy, energy[idx + idy * N]);
    }
}

/*
*   energy is the function used to calculate the energy between
*   (center, up), (center, down), (center, left), (center, right)
*/
__device__ double local_energy(int up, int down, int left, int right, int center){
    return -center * (up + down + left + right);
}

__global__ void updateEnergy(int* lattice, double* energy, double* energy2, double* mag,double* mag2, int init){

    const unsigned int idx = blockIdx.x * blockDim.y + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int idx_l = (idx - 1 + N) % N;
    const unsigned int idx_r = (idx + 1 + N) % N;
    const unsigned int idy_u = (idy - 1 + N) % N;
    const unsigned int idy_d = (idy + 1 + N) % N;
    int up, down, left, right, center;
    double site_E;

    up = lattice[idx + idy_u * N];
    down = lattice[idx + idy_d * N];
    left = lattice[idx_l + idy * N];
    right = lattice[idx_r + idy * N];
    center = lattice[idx + idy * N];

    if (idx < N && idy < N){
        site_E = local_energy(up, down, left, right, center) / 2.0;

        if(init == 1){
            energy[idx + N * idy] = 1.0 * site_E / (TIME_LENGTH + 1);
            energy2[idx + N * idy] = 1.0 * site_E * site_E / (TIME_LENGTH + 1);
            mag[idx + N * idy] = 1.0 * center / (TIME_LENGTH + 1);
            mag2[idx + N * idy] = 1.0 * center * center / (TIME_LENGTH + 1);
        }
        else{
            energy[idx + N * idy] += 1.0 * site_E / (TIME_LENGTH + 1);
            energy2[idx + N * idy] += 1.0 * site_E * site_E / (TIME_LENGTH + 1);
            mag[idx + N * idy] += 1.0 * center / (TIME_LENGTH + 1);
            mag2[idx + N * idy] += 1.0 * center * center / (TIME_LENGTH + 1);
        }
    }
}

/*
*   Commandline inputs option
*   1. Tempurature (T)
*
*/
int main (int argc, char *argv[]){

    int *lattice;
    int *d_lattice;

    double *energy;
    double *d_energy;

    double *energy2;
    double *d_energy2;

    double *mag;
    double *d_mag;

    double *mag2;
    double *d_mag2;

    hiprandState *d_states;

    double T = 2;
    int warmsteps = 1e4;
    int nout = TIME_LENGTH;
    // int warp = 1e3;

    int numthreadx = 16;
    int numthready = 16;
    int numblocksX = LATTICE_LENGTH / numthreadx;
    int numblocksY = LATTICE_LENGTH / numthready;

    // First input: Tempurature. Usually between (1, 6),
    // Critical Tempurature is around 2.2
    T = argc > 1 ? atof(argv[1]) : 2;

    // Define the size of lattice and energy
    const size_t bytes_int = LATTICE_2 * sizeof(int);
    const size_t bytes_double = LATTICE_2 * sizeof(double);

    // Allocate memory for lattice. It is a lattice^2 long array.
    // The value can only be 1 or -1.
    lattice = (int*)malloc(LATTICE_2 * sizeof(int));
    energy = (double*)malloc(LATTICE_2 * sizeof(double));
    energy2 = (double*)malloc(LATTICE_2 * sizeof(double));
    mag = (double*)malloc(LATTICE_2 * sizeof(double));
    mag2 = (double*)malloc(LATTICE_2 * sizeof(double));

    // initialize lattice by rand(-1, 1)
    for(int i = 0; i < LATTICE_2; i++){
        lattice[i] = 2 * (rand() % 2) - 1;
        energy[i] = 0.0;
        energy2[i] = 0.0;
        mag[i] = 0.0;
        mag2[i] = 0.0;
    }

    // Set dimensions of block and grid
    dim3 grid(numblocksX, numblocksY, 1);
    dim3 thread(numthreadx, numthready,1);

    // beta is a parameter in the probability
    double beta = 1.0 / BOLTZMANN_CONST / T;

    // Allocate memoery in device and copy from host to device
    hipMalloc((void **)&d_lattice, bytes_int);
    hipMalloc((void **)&d_energy, bytes_double);
    hipMalloc((void **)&d_energy2, bytes_double);
    hipMalloc((void **)&d_mag, bytes_double);
    hipMalloc((void **)&d_mag2, bytes_double);
    hipMalloc((void **)&d_states, LATTICE_2 * sizeof(hiprandState));

    hipMemcpy(d_lattice, lattice, bytes_int, hipMemcpyHostToDevice);
    hipMemcpy(d_energy, energy, bytes_double, hipMemcpyHostToDevice);
    hipMemcpy(d_energy2, energy2, bytes_double, hipMemcpyHostToDevice);
    hipMemcpy(d_mag, mag, bytes_double, hipMemcpyHostToDevice);
    hipMemcpy(d_mag2, mag2, bytes_double, hipMemcpyHostToDevice);

    // To change the buffer size of printf; otherwise it cannot print all data
    hipDeviceSetLimit(hipLimitPrintfFifoSize, N * N * sizeof(int) * N);

    ini_rng<<<grid, thread>>>(d_states, time(NULL));

    // Warmup process
    for (int iter = 0; iter < warmsteps; iter++){
        update<<<grid, thread>>>(d_lattice, 0, beta, d_states);
        update<<<grid, thread>>>(d_lattice, 1, beta, d_states);
        // cudaDeviceSynchronize();
    }
    updateEnergy<<<grid, thread>>>(d_lattice, d_energy, d_energy2, d_mag, d_mag2, 1);
    // Measure process
    for (int nstep = 0; nstep < nout; nstep++){
        update<<<grid, thread>>>(d_lattice, 0, beta, d_states);
        update<<<grid, thread>>>(d_lattice, 1, beta, d_states);
        updateEnergy<<<grid, thread>>>(d_lattice, d_energy, d_energy2, d_mag, d_mag2, 0);
    }
    // printstate<<<grid, thread>>>(d_energy);
    hipMemcpy(energy, d_energy, bytes_double, hipMemcpyDeviceToHost);
    hipMemcpy(energy2, d_energy2, bytes_double, hipMemcpyDeviceToHost);
    hipMemcpy(mag, d_mag, bytes_double, hipMemcpyDeviceToHost);
    hipMemcpy(mag2, d_mag2, bytes_double, hipMemcpyDeviceToHost);

    double sum_E = 0.0;
    double sum_E2 = 0.0;
    double sum_site = 0.0;
    double sum_site2 = 0.0;
    // double sum2 = 0.0;

    for (int i = 0; i < N ; i++){
        for (int j = 0; j < N; j++){
            sum_E += energy[i + j * N];
            sum_E2 += energy2[i + j * N];
            sum_site += mag[i + j * N];
            sum_site2 += mag2[i + j * N];
        }
    }

    double aver_E = 1.0 * sum_E / LATTICE_2;
    double aver_E2 = 1.0 * sum_E2 / LATTICE_2;
    double aver_site = 1.0 * sum_site / LATTICE_2;
    double aver_site2 = 1.0 * sum_site2 / LATTICE_2;

    double heat_capacity = 1.0 * (aver_E2 - aver_E * aver_E) / T / T;
    double mag_sus = 1.0 * (aver_site2 - aver_site * aver_site) / T;

    printf("%f\n", T);
    printf("%d\n", LATTICE_LENGTH);
    printf("%f\n", aver_E);
    printf("%f\n", heat_capacity);
    printf("%f\n", fabs(aver_site));
    printf("%f\n", mag_sus );

    //
    // printf("%s\n", );
    // printf("%f\n", 0.5 * sum / LATTICE_2);
    // printstate<<<grid, thread>>>(d_energy);

    free(lattice);
    hipFree(d_lattice);

    free(energy);
    hipFree(d_energy);

    free(energy2);
    hipFree(d_energy2);

    free(mag);
    hipFree(d_mag);

    free(mag2);
    hipFree(d_mag2);

}
